#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <dace/dace.h>


struct add3s2_t {
    dace::cuda::Context *gpu_context;
};



DACE_EXPORTED int __dace_init_cuda(add3s2_t *__state, int NE);
DACE_EXPORTED void __dace_exit_cuda(add3s2_t *__state);



int __dace_init_cuda(add3s2_t *__state, int NE) {
    int count;

    // Check that we are able to run cuda code
    if (hipGetDeviceCount(&count) != hipSuccess)
    {
        printf("ERROR: GPU drivers are not configured or cuda-capable device "
               "not found\n");
        return 1;
    }
    if (count == 0)
    {
        printf("ERROR: No cuda-capable devices found\n");
        return 2;
    }

    // Initialize cuda before we run the application
    float *dev_X;
    hipMalloc((void **) &dev_X, 1);
    hipFree(dev_X);

    

    __state->gpu_context = new dace::cuda::Context(1, 1);

    // Create cuda streams and events
    for(int i = 0; i < 1; ++i) {
        hipStreamCreateWithFlags(&__state->gpu_context->streams[i], hipStreamNonBlocking);
    }
    for(int i = 0; i < 1; ++i) {
        hipEventCreateWithFlags(&__state->gpu_context->events[i], hipEventDisableTiming);
    }

    

    return 0;
}

void __dace_exit_cuda(add3s2_t *__state) {
    

    // Destroy cuda streams and events
    for(int i = 0; i < 1; ++i) {
        hipStreamDestroy(__state->gpu_context->streams[i]);
    }
    for(int i = 0; i < 1; ++i) {
        hipEventDestroy(__state->gpu_context->events[i]);
    }

    delete __state->gpu_context;
}

__global__ void add3s2_18_0_0_0(double * __restrict__ a, const double * __restrict__ b, const double * __restrict__ c, int NE, const double c1, const double c2) {
    {
        int i = (blockIdx.x * 32 + threadIdx.x);
        double __tmp1;
        double __tmp2;
        double __tmp3;
        if (i < NE) {
            {
                double __in1 = c1;
                double __in2 = b[i];
                double __out;

                ///////////////////
                // Tasklet code (_Mult_)
                __out = (__in1 * __in2);
                ///////////////////

                __tmp1 = __out;
            }
            {
                double __in1 = c2;
                double __in2 = c[i];
                double __out;

                ///////////////////
                // Tasklet code (_Mult_)
                __out = (__in1 * __in2);
                ///////////////////

                __tmp2 = __out;
            }
            {
                double __in2 = __tmp2;
                double __in1 = __tmp1;
                double __out;

                ///////////////////
                // Tasklet code (_Add_)
                __out = (__in1 + __in2);
                ///////////////////

                __tmp3 = __out;
            }
            {
                double __inp = __tmp3;
                double __out;

                ///////////////////
                // Tasklet code (assign_19_8)
                __out = __inp;
                ///////////////////

                a[i] = __out;
            }
        }
    }
}


DACE_EXPORTED void __dace_runkernel_add3s2_18_0_0_0(add3s2_t *__state, double * __restrict__ a, const double * __restrict__ b, const double * __restrict__ c, int NE, const double c1, const double c2);
void __dace_runkernel_add3s2_18_0_0_0(add3s2_t *__state, double * __restrict__ a, const double * __restrict__ b, const double * __restrict__ c, int NE, const double c1, const double c2)
{

    void  *add3s2_18_0_0_0_args[] = { (void *)&a, (void *)&b, (void *)&c, (void *)&NE, (void *)&c1, (void *)&c2 };
    hipLaunchKernel((void*)add3s2_18_0_0_0, dim3(int_ceil(int_ceil(NE, 1), 32), 1, 1), dim3(32, 1, 1), add3s2_18_0_0_0_args, 0, __state->gpu_context->streams[0]);
}

